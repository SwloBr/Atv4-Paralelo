#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#define NTHREADS 256

// Função de kernel para calcular a contagem dos dígitos
__global__ void countDigitsKernel(int *d_arr, int *d_count, int n, int exp) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int digit = (d_arr[idx] / exp) % 10;
        atomicAdd(&d_count[digit], 1);
    }
}

// Função de kernel para calcular a posição dos elementos (usando a contagem)
__global__ void prefixSumKernel(int *d_count) {
    for (int i = 1; i < 10; ++i) {
        d_count[i] += d_count[i - 1];
    }
}

// Função de kernel para ordenar os elementos
__global__ void reorderElementsKernel(int *d_arr, int *d_output, int *d_count, int n, int exp) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int digit = (d_arr[idx] / exp) % 10;
        int pos = atomicSub(&d_count[digit], 1) - 1;
        d_output[pos] = d_arr[idx];
    }
}

// Função principal Radix Sort usando CUDA
void radixSortCUDA(std::vector<int>& arr) {
    int n = arr.size();
    int *d_arr, *d_output, *d_count;

    // Alocar memória na GPU
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));
    hipMalloc(&d_count, 10 * sizeof(int));

    // Copiar os dados para a GPU
    hipMemcpy(d_arr, arr.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Encontrar o maior valor para determinar o número de dígitos
    int maxVal = *std::max_element(arr.begin(), arr.end());

    // Para cada dígito (unidade, dezena, centena, etc.)
    for (int exp = 1; maxVal / exp > 0; exp *= 10) {
        // Inicializar contagem
        hipMemset(d_count, 0, 10 * sizeof(int));

        // Configurar número de blocos e threads
        int nBlocks = (n + NTHREADS - 1) / NTHREADS;

        // Contar os dígitos (Counting Sort)
        countDigitsKernel<<<nBlocks, NTHREADS>>>(d_arr, d_count, n, exp);
        hipDeviceSynchronize();

        // Calcular prefix sum (exclusivo)
        prefixSumKernel<<<1, 1>>>(d_count);
        hipDeviceSynchronize();

        // Reordenar os elementos
        reorderElementsKernel<<<nBlocks, NTHREADS>>>(d_arr, d_output, d_count, n, exp);
        hipDeviceSynchronize();

        // Copiar o resultado temporário de volta para d_arr para a próxima iteração
        hipMemcpy(d_arr, d_output, n * sizeof(int), hipMemcpyDeviceToDevice);
    }

    // Copiar o resultado de volta para o host
    hipMemcpy(arr.data(), d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memória na GPU
    hipFree(d_arr);
    hipFree(d_output);
    hipFree(d_count);
}

int main() {
    // Tamanhos dos arrays a serem testados
    std::vector<int> sizes = {100, 1000, 10000, 100000, 1000000};

    for (int size : sizes) {
        std::vector<int> arr(size);

        // Preencher o vetor com valores aleatórios
        srand(time(0));
        for (int i = 0; i < size; i++) {
            arr[i] = rand() % size;
        }

        // Medir o tempo de execução
        clock_t start = clock();
        radixSortCUDA(arr);
        clock_t end = clock();

        double elapsed = double(end - start) / CLOCKS_PER_SEC;
        std::cout << "Array de tamanho " << size << " ordenado em " << elapsed << " segundos (CUDA paralelo)." << std::endl;
    }

    return 0;
}
